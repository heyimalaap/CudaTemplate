#include "hip/hip_runtime.h"
#include <thrust/execution_policy.h>
#include <thrust/device_vector.h>
#include <cub/block/block_reduce.cuh>
#include <cuda/atomic>
#include <cuda/cmath>
#include <cuda/std/span>
#include <cstdio>

template <int block_size>
__global__ void reduce(cuda::std::span<int const> data, cuda::std::span<int> result) {
  using BlockReduce = hipcub::BlockReduce<int, block_size>;
  __shared__ typename BlockReduce::TempStorage temp_storage;

  int const index = threadIdx.x + blockIdx.x * blockDim.x;
  int sum = 0;
  if (index < data.size()) {
    sum += data[index];
  }
  sum = BlockReduce(temp_storage).Sum(sum);

  if (threadIdx.x == 0) {
    cuda::atomic_ref<int, cuda::thread_scope_device> atomic_result(result.front());
    atomic_result.fetch_add(sum, cuda::memory_order_relaxed);
  }
}

int main() {

  // Allocate and initialize input data
  int const N = 1000;
  thrust::device_vector<int> data(N);
  thrust::fill(data.begin(), data.end(), 1);
  
  // Allocate output data
  thrust::device_vector<int> kernel_result(1);

  // Compute the sum reduction of `data` using a custom kernel
  constexpr int block_size = 256;
  int const num_blocks = cuda::ceil_div(N, block_size);
  reduce<block_size><<<num_blocks, block_size>>>(cuda::std::span<int const>(thrust::raw_pointer_cast(data.data()), data.size()),
                                                 cuda::std::span<int>(thrust::raw_pointer_cast(kernel_result.data()), 1));

  auto const err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    std::cout << "Error: " << hipGetErrorString(err) << std::endl;
    return -1;
  }

  int const custom_result = kernel_result[0];

  // Compute the same sum reduction using Thrust
  int const thrust_result = thrust::reduce(thrust::device, data.begin(), data.end(), 0);

  // Ensure the two solutions are identical
  std::printf("Custom kernel sum: %d\n", custom_result);
  std::printf("Thrust reduce sum: %d\n", thrust_result);
  assert(kernel_result[0] == thrust_result);
  return 0;
}